#include "hip/hip_runtime.h"
// cuda_sum.cu
#include "cuda_hello.h"
#include <iostream>

__global__ void arraySumKernel(const int* array1, const int* array2, int* result, size_t size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        result[tid] = array1[tid] + array2[tid];
    }
}

void arraySumCUDA(const std::vector<int>& array1, const std::vector<int>& array2, std::vector<int>& result) {
    size_t size = array1.size();
    int* d_array1, * d_array2, * d_result;

    // Allocate memory on the device
    hipMalloc(&d_array1, size * sizeof(int));
    hipMalloc(&d_array2, size * sizeof(int));
    hipMalloc(&d_result, size * sizeof(int));

    // Copy input arrays to the device
    hipMemcpy(d_array1, array1.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_array2, array2.data(), size * sizeof(int), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    arraySumKernel<<<(size + 255) / 256, 256>>>(d_array1, d_array2, d_result, size);

    // Copy the result back to the host
    hipMemcpy(result.data(), d_result, size * sizeof(int), hipMemcpyDeviceToHost);

    //     // Print the result on the host side
    // std::cout << "Result from CUDA: ";
    // for (int i = 0; i < size; ++i) {
    //     std::cout << result[i] << " ";
    // }
    // std::cout << std::endl;

    // Free device memory
    hipFree(d_array1);
    hipFree(d_array2);
    hipFree(d_result);
}
